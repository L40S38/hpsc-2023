#include "hip/hip_runtime.h"
//#pragma execution_character_set("utf-8")

#include <iostream>
#include <vector>
#include <cmath>
#include <cstdio>
#include <cstdlib>
#include <stdio.h>
#include <chrono>

// グローバル変数
const int nx = 41;
const int ny = 41;
const int nt = 500;
//const int nt = 10;
const int nit = 50;
const double dx = 2.0 / (nx - 1);
const double dy = 2.0 / (ny - 1);
const double dt = 0.01;
const double rho = 1.0;
const double nu = 0.02;

double *u, *v, *p, *b, *pn, *un, *vn;

#define u(y,x) u[(y)*ny + (x)]
#define v(y,x) v[(y)*ny + (x)]
#define p(y,x) p[(y)*ny + (x)]
#define b(y,x) b[(y)*ny + (x)]
#define pn(y,x) pn[(y)*ny + (x)]
#define un(y,x) un[(y)*ny + (x)]
#define vn(y,x) vn[(y)*ny + (x)]

#define a(y,x) a[y*ny+x]
#define an(y,x) an[y*ny+x]

// プロットデータをファイルに保存する関数
void sendDataPressure(const std::vector<double>& x, const std::vector<double>& y,
                        //const std::vector<std::vector<double>>& p, 
                        double *p, FILE* fp) {
    //FILE* fp = fopen(filename.c_str(), "w", "utf-8");
    if (fp != nullptr) {
        for (int i = 0; i < ny; i++) {
            for (int j = 0; j < nx; j++) {
                fprintf(fp, "%f %f %f\n", x[j], y[i], p(i,j));
            }
            //fprintf(fp,"\n");
        }
        fprintf(fp, "e\n");
        //fclose(fp);
    } else {
        //std::cerr << "Failed to open file: " << filename << std::endl;
        std::cerr << "Failed to open file" << std::endl;
    }
}

void sendDataVerocity(const std::vector<double>& x, const std::vector<double>& y,
                        //const std::vector<std::vector<double>>& u, const std::vector<std::vector<double>>& v,
                        double *u, double *v, FILE* fp) {
    //FILE* fp = fopen(filename.c_str(), "w");
    if (fp != nullptr) {
        for (int i = 1; i < ny; i = i+2) {
            for (int j = 1; j < nx; j = j+2) {
                fprintf(fp, "%f %f %f %f\n", x[j], y[i], u(i,j)*0.5, v(i,j)*0.5);
            }
        }
        fprintf(fp, "e\n");
        //fclose(fp);
    } else {
        std::cerr << "Failed to open file" << std::endl;
    }
}

//境界条件
void border(double *u, double *v) {
    for (int i = 0; i < nx; i++) {
        u(0,i) = 0.0;
        u(ny - 1,i) = 1.0;
        v(0,i) = 0.0;
        v(ny - 1,i) = 0.0;
    }
    for (int i = 0; i < ny; i++) {
        u(i,0) = 0.0;
        u(i,nx - 1) = 0.0;
        v(i,0) = 0.0;
        v(i,nx - 1) = 0.0;
    }
}

// 初期条件の設定
/*
void initialize(std::vector<std::vector<double>>& u, std::vector<std::vector<double>>& v,
                std::vector<std::vector<double>>& p, std::vector<std::vector<double>>& b) {
    for (int i = 0; i < ny; i++) {
        for (int j = 0; j < nx; j++) {
            u[i][j] = 0.0;
            v[i][j] = 0.0;
            p[i][j] = 0.0;
            b[i][j] = 0.0;
        }
    }
}
*/

__global__ void init_zero(double *u, double *v, double *p, double *b){
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    u[i] = 0.0;
    v[i] = 0.0;
    p[i] = 0.0;
    b[i] = 0.0;
}

void free_memories(double *u, double *v, double *p, double *b, 
                            double *pn, double *un, double *vn){
    hipFree(u);
    hipFree(v);
    hipFree(p);
    hipFree(b);
    hipFree(pn);
    hipFree(un);
    hipFree(vn);
}

__global__ void data_insert_b(double *b, double *u, double *v){
    int i = blockIdx.x+1;
    int j = threadIdx.x+1;
    //if(i>0 && i<nx-1 && j>0 && j<ny-1){
    b(j,i) = rho * (1 / dt *
            ((u(j,i+1) - u(j,i-1)) / (2 * dx) + (v(j+1,i) - v(j-1,i)) / (2 * dy)) -
            ((u(j,i+1) - u(j,i-1)) / (2 * dx)) * ((u(j,i+1) - u(j,i-1)) / (2 * dx)) -
            2 * ((u(j+1,i) - u(j-1,i)) / (2 * dy)) *
                ((v(j,i+1) - v(j,i-1)) / (2 * dx)) -
            ((v(j+1,i) - v(j-1,i)) / (2 * dy)) * ((v(j+1,i) - v(j-1,i)) / (2 * dy)));
    //}
}

__global__ void copy_array(double *an, double *a){
    int i = blockIdx.x;
    int j = threadIdx.x;

    an(j,i) = a(j,i);
}

__global__ void data_insert_p(double *p, double *b){
    int i = blockIdx.x+1;
    int j = threadIdx.x+1;
    extern __shared__ double pn[];
    __syncthreads();
    pn(j,i) = p(i);
    __syncthreads();
    //if(i>0 && i<nx-1 && j>0 && j<ny-1){
    p(j,i) = (dy * dy * (pn(j,i+1) + pn(j,i-1)) +
                dx * dx * (pn(j+1,i) + pn(j-1,i)) -
                b(j,i) * dx * dx * dy * dy) /
                (2 * (dx * dx + dy * dy));
    //}
}

__global__ void data_insert_u_v(double *u, double *v, double *un, double *vn, double *p){
    int i = blockIdx.x+1;
    int j = threadIdx.x+1;
    //if(i>0 && i<nx-1 && j>0 && j<ny-1){
    u(j,i) = un(j,i) - un(j,i) * dt / dx * (un(j,i) - un(j,i-1)) -
                                    un(j,i) * dt / dy * (un(j,i) - un(j-1,i)) -
                                    dt / (2 * rho * dx) * (p(j,i+1) - p(j,i-1)) +
                                    nu * dt / (dx * dx) * (un(j,i+1) - 2 * un(j,i) + un(j,i-1)) +
                                    nu * dt / (dy * dy) * (un(j+1,i) - 2 * un(j,i) + un(j-1,i));
    v(j,i) = vn(j,i) - vn(j,i) * dt / dx * (vn(j,i) - vn(j,i-1)) -
                                    vn(j,i) * dt / dy * (vn(j,i) - vn(j-1,i)) -
                                    dt / (2 * rho * dx) * (p(j+1,i) - p(j-1,i)) +
                                    nu * dt / (dx * dx) * (vn(j,i+1) - 2 * vn(j,i) + vn(j,i-1)) +
                                    nu * dt / (dy * dy) * (vn(j+1,i) - 2 * vn(j,i) + vn(j-1,i));
    //}
}
/*
__global__ void data_insert_v(double *v, double *un, double *vn, double *p){
    int i = blockIdx.x+1;
    int j = threadIdx.x+1;
    //if(i>0 && i<nx-1 && j>0 && j<ny-1){
    v(j,i) = vn(j,i) - vn(j,i) * dt / dx * (vn(j,i) - vn(j,i-1)) -
                                    vn(j,i) * dt / dy * (vn(j,i) - vn(j-1,i)) -
                                    dt / (2 * rho * dx) * (p(j+1,i) - p(j-1,i)) +
                                    nu * dt / (dx * dx) * (vn(j,i+1) - 2 * vn(j,i) + vn(j,i-1)) +
                                    nu * dt / (dy * dy) * (vn(j+1,i) - 2 * vn(j,i) + vn(j-1,i));
    //}
}
*/

int main(void){
    // x軸とy軸
    std::vector<double> x(nx);
    std::vector<double> y(ny);
    for (int i = 0; i < nx; i++) {
        x[i] = i * dx;
    }
    for (int i = 0; i < ny; i++) {
        y[i] = i * dy;
    }

    // 変数の初期化
    /*
    std::vector<std::vector<double>> u(ny, std::vector<double>(nx));
    std::vector<std::vector<double>> v(ny, std::vector<double>(nx));
    std::vector<std::vector<double>> p(ny, std::vector<double>(nx));
    std::vector<std::vector<double>> b(ny, std::vector<double>(nx));
    */
    //initialize(u, v, p, b, pn, un, vn);
    hipMallocManaged(&u, nx*ny*sizeof(double));
    hipMallocManaged(&v, nx*ny*sizeof(double));
    hipMallocManaged(&p, nx*ny*sizeof(double));
    hipMallocManaged(&b, nx*ny*sizeof(double));
    hipMallocManaged(&pn, nx*ny*sizeof(double));
    hipMallocManaged(&un, nx*ny*sizeof(double));
    hipMallocManaged(&vn, nx*ny*sizeof(double));
    init_zero<<<nx,ny>>>(u,v,p,b);
    hipDeviceSynchronize();
    std::chrono::steady_clock::time_point tic, toc;
    double time;

    // gnuplotのパイプラインの作成
    FILE* gnuplotPipe = popen("gnuplot -persist", "w");
    if (gnuplotPipe == nullptr) {
        std::cerr << "Failed to open gnuplot pipe." << std::endl;
        return 1;
    } else {
        // git animate settings
        fprintf(gnuplotPipe, "reset\n");
        //fprintf(gnuplotPipe, "set terminal gif animate\n");
        //fprintf(gnuplotPipe, "set output '10_cavity.gif'\n");
    }

    for(int n=0; n<nt; n++){
        //タイム計測
        toc = std::chrono::steady_clock::now();
        /*
        for(int j=1; j<ny-1; j++){
            for(int i=1; i<nx-1; i++){
                b[j][i] = rho * (1 / dt *
                                 ((u[j][i+1] - u[j][i-1]) / (2 * dx) + (v[j+1][i] - v[j-1][i]) / (2 * dy)) -
                                 ((u[j][i+1] - u[j][i-1]) / (2 * dx)) * ((u[j][i+1] - u[j][i-1]) / (2 * dx)) -
                                 2 * ((u[j+1][i] - u[j-1][i]) / (2 * dy)) *
                                     ((v[j][i+1] - v[j][i-1]) / (2 * dx)) -
                                 ((v[j+1][i] - v[j-1][i]) / (2 * dy)) * ((v[j+1][i] - v[j-1][i]) / (2 * dy)));
            }
        }
        */
        data_insert_b<<<nx-2,ny-2>>>(b,u,v);
        hipDeviceSynchronize();
        for(int it=0; it<nit; it++){
            //std::vector<std::vector<double>> pn = p;
            //copy_array<<<nx,ny>>>(pn,p);
            //hipDeviceSynchronize();
            /*
            for(int j=1; j<ny-1; j++){
                for(int i=1; i<nx-1; i++){
                    p[j][i] =
                        (dy * dy * (pn[j][i+1] + pn[j][i-1]) +
                         dx * dx * (pn[j+1][i] + pn[j-1][i]) -
                         b[j][i] * dx * dx * dy * dy) /
                        (2 * (dx * dx + dy * dy));
                }
            }*/
            data_insert_p<<<nx-2,ny-2>>>(p,b);
            hipDeviceSynchronize();
            for (int i = 0; i < nx; i++) {
                p(0,i) = p(1,i);
                p(ny - 1,i) = 0.0;
            }
            for (int i = 0; i < ny; i++) {
                p(i,0) = p(i,1);
                p(i,nx - 1) = p(i,nx - 2);
            }
        }
        //std::vector<std::vector<double>> un = u;
        //std::vector<std::vector<double>> vn = v;
        copy_array<<<nx,ny>>>(un,u);
        copy_array<<<nx,ny>>>(vn,v);
        hipDeviceSynchronize();
        /*
        for (int j=1; j<ny-1; j++) {
            for (int i=1; i<nx-1; i++) {
                u[j][i] = un[j][i] - un[j][i] * dt / dx * (un[j][i] - un[j][i-1]) -
                                    un[j][i] * dt / dy * (un[j][i] - un[j-1][i]) -
                                    dt / (2 * rho * dx) * (p[j][i+1] - p[j][i-1]) +
                                    nu * dt / (dx * dx) * (un[j][i+1] - 2 * un[j][i] + un[j][i-1]) +
                                    nu * dt / (dy * dy) * (un[j+1][i] - 2 * un[j][i] + un[j-1][i]);
                v[j][i] = vn[j][i] - vn[j][i] * dt / dx * (vn[j][i] - vn[j][i-1]) -
                                    vn[j][i] * dt / dy * (vn[j][i] - vn[j-1][i]) -
                                    dt / (2 * rho * dx) * (p[j+1][i] - p[j-1][i]) +
                                    nu * dt / (dx * dx) * (vn[j][i+1] - 2 * vn[j][i] + vn[j][i-1]) +
                                    nu * dt / (dy * dy) * (vn[j+1][i] - 2 * vn[j][i] + vn[j-1][i]);
            }
        }
        */
        data_insert_u_v<<<nx-2,ny-2>>>(u,v,un,vn,p);
        //data_insert_v<<<nx-2,ny-2>>>(v,un,vn,p);
        hipDeviceSynchronize();
        border(u,v);

        // 時間計測, gnuplotの部分は除外
        tic = std::chrono::steady_clock::now();
        time = std::chrono::duration<double>(tic-toc).count();
        std::cout << n + 1 << "," << time << std::endl;

        // gnuplotにプロットコマンドを送信
        fprintf(gnuplotPipe, "set title 'Pressure'\n");
        fprintf(gnuplotPipe, "set xrange [0:2]\n");
        fprintf(gnuplotPipe, "set xlabel 'x'\n");
        fprintf(gnuplotPipe, "set yrange [0:2]\n");
        fprintf(gnuplotPipe, "set ylabel 'y'\n");
        fprintf(gnuplotPipe, "set cbrange [-0.6:0.6]\n");
        fprintf(gnuplotPipe, "set palette maxcolors 24\n");
        fprintf(gnuplotPipe, "set palette defined (-0.6 'blue', 0 'white', 0.6 'red')\n");
        fprintf(gnuplotPipe, "set nokey\n");

        fprintf(gnuplotPipe, "plot '-' u 1:2:3 with image, '-' with vector lc black\n");
        sendDataPressure(x, y, p, gnuplotPipe);
        sendDataVerocity(x, y, u, v, gnuplotPipe);
        fprintf(gnuplotPipe, "pause .01\n");
        fprintf(gnuplotPipe, "\n\n");
        fflush(gnuplotPipe);
    }

    // gnuplotパイプラインを閉じる
    pclose(gnuplotPipe);

    //メモリを解放する
    free_memories(u, v, p, b, pn, un, vn);
    return 0;
}