
#include <hip/hip_runtime.h>
#include <cstdio>
#include <cstdlib>
#include <vector>
__global__ void zero_init(int *array){/*ゼロ初期化*/
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  array[i] = 0;
  __syncthreads();
}

__global__ void reduction_add(int *array,int *key){/*加算のためのreduction*/
  int i = threadIdx.x;
  int j = blockIdx.x;
  if(key[i]==j)atomicAdd(&array[j],1);
}

__global__ void scan(int *a, int *b, int range) {/*prefix sumのための関数*/
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  for(int j=1; j<range; j<<=1) {
    b[i] = a[i];
    __syncthreads();
    if(i>=j) a[i] += b[i-j];
    __syncthreads();
  }
  b[i] = a[i];
  __syncthreads();
  if(i==0)a[0]=0;
  else a[i]=b[i-1];
}

__global__ void sort(int *offset, int *key, int N) {/*bucket sort*/
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  int begin,end;
  if(i==0)begin=0;
  else begin=offset[i-1];
  if(i==N-1)end=N;
  else end=offset[i];

  for(int j=begin;j<end;j++) {
    key[j]=i;
  }
}

int main() {
  int n = 50;
  int range = 5;
  int *key,*bucket,*offset;
  /*メモリ確保*/
  hipMallocManaged(&key, n*sizeof(int));
  hipMallocManaged(&bucket, range*sizeof(int));
  hipMallocManaged(&offset, range*sizeof(int));
  for (int i=0; i<n; i++) {
    key[i] = rand() % range;
    printf("%d ",key[i]);
  }
  printf("\n");
 
  zero_init<<<1,range>>>(bucket);
  zero_init<<<1,range>>>(offset);
  hipDeviceSynchronize();
  reduction_add<<<range,n>>>(bucket,key);
  hipDeviceSynchronize();
  scan<<<1,range>>>(bucket,offset,range);
  hipDeviceSynchronize();
  /*
  for (int i=0; i<range; i++) {
    printf("%d ",bucket[i]);
  }
  printf("\n");
  for (int i=0; i<range; i++) {
    printf("%d ",offset[i]);
  }
  printf("\n");
  */
  sort<<<1,range>>>(offset,key,n);
  hipDeviceSynchronize();
  for (int i=0; i<n; i++) {
    printf("%d ",key[i]);
  }
  printf("\n");
}
